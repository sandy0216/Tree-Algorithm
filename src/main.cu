#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <vector>
#include "../inc/init.h"
#include "../inc/def_node.h"
#include "../inc/create_tree.h"
#include "../inc/force.h"
#include "../inc/print_tree.h"
#include "../inc/tool_main.h"

using namespace std;


int main( int argc, char* argv[] )
{
	double *x, *y, *mass;
	double *vx, *vy;
	double *fx,*fy;
	double *V;
	double E,Ek;
	double boxsize = 100.0;
	double region = 20.0;  // restrict position of the initial particles
	double maxmass = 100.0;
	double theta = 0.8;

	double dt = 1e-5;
	double endtime = 5e-5;

	int n=100;

	x = (double *)malloc(n*sizeof(double));
	y = (double *)malloc(n*sizeof(double));
	mass = (double *)malloc(n*sizeof(double));
	vx = (double *)malloc(n*sizeof(double));
	vy = (double *)malloc(n*sizeof(double));
	fx = (double *)malloc(n*sizeof(double));
	fy = (double *)malloc(n*sizeof(double));
	V  = (double *)malloc(n*sizeof(double));

	//==================initial conditions========================
	// Create initial conditions
	init(x, y, mass, n, region, boxsize, maxmass);
	for( int i=0;i<n;i++ ){
		vx[i]=vy[i]=0.0;
		//mass[i]=10;
	}
	
	// Record the initial conditions
	FILE *initfile;
	initfile = fopen("./input/init.dat","w");
	fprintf(initfile, "index\tx\ty\tmass\n");
	for( int i=0;i<n;i++ ){
		fprintf(initfile, "%d\t%.3f\t%.3f\t%.3f\n",i,x[i],y[i],mass[i]);
	}
	fclose(initfile);
	// End of creating intial conditions
	
	//=================Evolution===============================
	double t=0.0;
	int step=0;
	int file=0;
	
	char preffix[15] = "./output/snap_";
	char number[5];
	char suffix[5] = ".dat";
	int length;
	
	while( t<endtime ){
		// Create tree
		NODE *head = new NODE();
		create_tree(head, x, y, mass, boxsize,n);
		//printf("End creating tree...\n");

		// Calculate force for each particles
		force(head, x, y, mass, fx, fy, theta,n);
		//printf("Finish calculating force...\n");

		update(x,y,vx,vy,n,dt);
		update(vx,vy,fx,fy,n,dt);
		potential(head,x,y,mass,V,theta,n);

		// Verification
		//if( step%10==0 ){
			printf("[Step %d] T=%.3e\n",step,t);
			E = 0;
			Ek = 0;
			for( int i=0;i<n;i++ ){
				E += V[i]+0.5*mass[i]*(pow(vx[i],2)+pow(vy[i],2));
				Ek += 0.5*mass[i]*(pow(vx[i],2)+pow(vy[i],2));
			}
			printf("Particle number remains:%d\n",n);
			printf("Energy conservation:%.3e\n",E);
			printf("Kinetic energy:%.3e\n",Ek);
		//}
	
		if( step%1000==0 ){
			//Output snapshots
			sprintf(number,"%d",file);
			length = snprintf(NULL, 0, "%s%s%s",preffix,number,suffix);
			char concated[length+1];
			snprintf(concated,sizeof(concated),"%s%s%s",preffix,number,suffix);
			FILE *outfile;
			outfile = fopen(concated,"w");
			fprintf(outfile, "index\tx\ty\n");
			for( int i=0;i<n;i++ ){
				fprintf(outfile, "%d\t%.3f\t%.3f\n",i,x[i],y[i]);
			}
			printf("Record position ...\n");
			fclose(outfile);
			file += 1;
		}

		// Move to next step
		t = t+dt;
		step = step+1;
	}
	

	return 0;
}
