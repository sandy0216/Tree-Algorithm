#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <vector>
#include <omp.h>
#include "../inc/init.h"
#include "../inc/def_node.h"
#include "../inc/create_tree.h"
#include "../inc/tool_gpu.h"
#include "../inc/force.h"
#include "../inc/print_tree.h"
#include "../inc/tool_main.h"
#include "../inc/param.h"
#include "../inc/heap.h"
#include "../inc/cuapi.h"
#include "../inc/merge_tree_gpu.h"
#include "../inc/force_gpu.h"
#include "../inc/tree_gpu.h"

using namespace std;

__device__ double d_boxsize;
__device__ double d_theta;
__device__ double d_eplison;
__device__ double d_dt;
__device__ int d_n_work;
__device__ int d_bx;
__device__ int d_n_thread;
__device__ int d_side;// =nx=ny
__device__ int share_node;
__device__ int global_node;

double boxsize,theta,dt,eplison;
unsigned long initial_n;
int nx,ny,tx,ty,bx,by;
int n_work,n_thread;
double endtime;
int endstep,recstep;

/*double boxsize = 100;
double theta   = 0.8;
double dt      = 1e-9;
unsigned long initial_n=1e7;
int nx=1024;
int ny=1024;
int tx=16;
int ty=16;
int bx=32;
int by=32;
int n_work=nx*ny;
int n_thread=tx*ty*bx*by;
double eplison = 3*boxsize/initial_n;*/

int main( int argc, char* argv[] )
{
	double *x, *y, *mass;
	double *vx, *vy;
	double *fx,*fy;
	double *V;
	double region = 100.0;  // restrict position of the initial particles
	double maxmass = 100.0;

	char buffer[50];
	if( strcmp(argv[1],"-CPU") && strcmp(argv[1],"-GPU") ){
		printf("Error input!!!\n");
		printf("Usage : ./tree -CPU/-GPU ./params\n");
		printf("%s\n",argv[1]);
		exit(1);
	}
	// Read the input file
	printf("=====Input parameters=====\n");
	FILE *params;
	params = fopen(argv[2],"r");
	fscanf(params,"%s%lf",buffer,&boxsize);
	printf("Boxsize \t\t= %.2f\n",boxsize);
	fscanf(params,"%s%d",buffer,&initial_n);
	printf("Number of particle \t= %d\n",initial_n);
	fscanf(params,"%s%lf",buffer,&theta);
	printf("Theta \t\t\t= %.2f\n",theta);
	fscanf(params,"%s%lf\n",buffer,&dt);
	printf("Timestep \t\t= %.2e\n",dt);
	fscanf(params,"%s%lf",buffer,&endtime);
	printf("End time \t\t= %.2e\n",endtime);
	fscanf(params,"%s%d",buffer,&endstep);
	printf("End step \t\t= %d\n",endstep);
	fscanf(params,"%s%d",buffer,&recstep);
	printf("Record data every %d steps\n",recstep);

	if( !strcmp(argv[1],"-GPU") ){
	fscanf(params,"%s%d\n",buffer,&nx);
	fscanf(params,"%s%d\n",buffer,&ny);
	printf("Number of subgrid \t= (%d,%d)\n",nx,ny);
	fscanf(params,"%s%d\n",buffer,&tx);
	fscanf(params,"%s%d\n",buffer,&ty);
	printf("Threads per block \t= (%d,%d)\n",tx,ty);
	fscanf(params,"%s%d\n",buffer,&bx);
	fscanf(params,"%s%d\n",buffer,&by);
	printf("Blocks per gird \t= (%d,%d)\n",bx,by);
	}
	printf("================================\n");
	n_work = nx*ny;
	n_thread = (tx*ty)*(bx*by);
	eplison = boxsize*3/initial_n;
	
	unsigned long    n  = initial_n;

	float testtime;
	float time = 0;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEvent_t tic,toc;
	hipEventCreate(&tic);
	hipEventCreate(&toc);

	x = (double *)malloc(n*sizeof(double));
	y = (double *)malloc(n*sizeof(double));
	mass = (double *)malloc(n*sizeof(double));
	vx = (double *)malloc(n*sizeof(double));
	vy = (double *)malloc(n*sizeof(double));
	fx = (double *)malloc(n*sizeof(double));
	fy = (double *)malloc(n*sizeof(double));
	V  = (double *)malloc(sizeof(double));

	//==================initial conditions========================
	// Create initial conditions
	init(x, y, mass, n, region, boxsize, maxmass);
	for( int i=0;i<n;i++ ){
		vx[i]=vy[i]=0.0;
		//mass[i]=10;
	}
	printf("Finsih creating initial condition...\n");
	
	// Record the initial conditions
#ifdef RECORD_INI
	FILE *initfile;
	initfile = fopen("./input/init.dat","w");
	fprintf(initfile, "index\tx\ty\tmass\n");
	for( int i=0;i<n;i++ ){
		fprintf(initfile, "%d\t%.3f\t%.3f\t%.3f\n",i,x[i],y[i],mass[i]);
	}
	fclose(initfile);
#endif
	// End of creating intial conditions
	
	//================================================================================//
	//		The following code is for GPU parallelization			  //
	//================================================================================//

	if( !strcmp(argv[1],"-GPU") ){
	hipEventRecord(tic,0);
	double t = 0.0;
	int step = 0;

	while( t<endtime ){
	printf("[Step %d] T=%.3e\n",step,t);	
	//====================GPU blocks & gird settings====================
	unsigned long gpu_memory = 0;
	int gid = 0;
	if( hipSetDevice(gid) != hipSuccess ){
		printf("!!! Cannot select GPU \n");
		exit(1);
	}
	hipSetDevice(gid);

	if( tx*ty>1024 ){
		printf("Number of threads per block must < 1024!!\n");
		exit(0);
	}
	dim3 threads(tx,ty);
	if( bx>65535 || by>65535 ){
		printf("The grid size exceeds the limit!\n");
		exit(0);
	}
	dim3 blocks(bx,by);
	//====================End of blocks & grid settings==================

	//====================Set basic parameters of GPU====================
	// Set global parameter
	hipMemcpyToSymbol(HIP_SYMBOL( d_boxsize), &boxsize, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL( d_side), &nx, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL( d_n_work), &n_work, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL( d_n_thread), &n_thread, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL( d_bx), &bx, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL( d_theta), &theta, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL( d_eplison), &eplison, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL( d_dt),&dt, sizeof(double));
	gpu_memory += 4*sizeof(int)+4*sizeof(double);
	
	// Deliver the information of each particle to GPU
	int   *d_n;
	hipMalloc((void**)&d_n, sizeof(int));
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
	double *d_x,*d_y,*d_mass;
	hipMalloc((void**)&d_x, n*sizeof(double));
	hipMalloc((void**)&d_y, n*sizeof(double));
	hipMalloc((void**)&d_mass, n*sizeof(double));
	hipMemcpy(d_x, x, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, n*sizeof(double), hipMemcpyHostToDevice);
	gpu_memory += sizeof(int)+3*n*sizeof(double);
	// Allocate memory for force
	double *d_fx,*d_fy,*d_V,*d_Ek;
	hipMalloc((void**)&d_fx,n*sizeof(double));
	hipMalloc((void**)&d_fy,n*sizeof(double));
	hipMalloc((void**)&d_V,sizeof(double));
	hipMalloc((void**)&d_Ek,sizeof(double));
	gpu_memory += sizeof(double)*(2*n+2);
	//====================End of setting basic parameters=====================

	//====================Split the particle into different subregion=====================
	// Record the region index of each particle
	int *index, *d_index;   
	index = (int *)malloc(n*sizeof(int));
	hipMalloc((void**)&d_index, n*sizeof(int));
	gpu_memory += sizeof(int)*n;	
	
	// Call kernel function :
	// Input  : parameters, postion fo the particles
	// Output : region index of each particles, number of particles in each region
	hipEventRecord(start,0);
	split<<<blocks,threads>>>(d_x,d_y,d_index,d_n);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&testtime, start, stop);
#ifdef OUTPUT_DETAIL
	printf("[step1] %.3e(ms) : Split particles into subregions\n",testtime);
	printf("[step1] GPU memory usage : %d bytes\n",gpu_memory);
#endif
	time += testtime;
	//===================End of splitting particles into different subregion============
	

	//==================='Merge' particles in different subregion into a particle================
	hipEventRecord(start,0);
	// Load memory for each subregion
	double *rx, *ry, *rmass;
        double *d_rx, *d_ry, *d_rmass;
	int *rn;
        int *d_rn;
	rx = (double *)malloc(n_work*sizeof(double));
	ry = (double *)malloc(n_work*sizeof(double));
	rmass = (double *)malloc(n_work*sizeof(double));
	rn = (int *)malloc(n_work*sizeof(int));
	hipMalloc((void**)&d_rx, n_work*sizeof(double));
	hipMalloc((void**)&d_ry, n_work*sizeof(double));
	hipMalloc((void**)&d_rmass, n_work*sizeof(double));
	hipMalloc((void**)&d_rn, n_work*sizeof(int));
	gpu_memory+=(n_work*(3*sizeof(double)+sizeof(int)));
	for( int i=0;i<n_work;i++ ){
		rx[i] = 0;
		ry[i] = 0;
		rmass[i] = 0;
		rn[i] = 0;
	}
	hipMemcpy(d_rx,rx,n_work*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_ry,ry,n_work*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_rmass,rmass,n_work*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_rn,rn,n_work*sizeof(int),hipMemcpyHostToDevice);

	// Call kernel function:
	// Input:origin information of each particles, particle index, load of each region, total number of particle, 
	//	region index, load of each thread
	// Output:information of output 'particles', and how many number they contain.
	merge_bottom<<<blocks,threads>>>(d_x,d_y,d_mass,d_index,d_rx,d_ry,d_rmass,d_rn,d_n);
	merge_bottom2<<<blocks,threads>>>(d_rx,d_ry,d_rmass);
	
	hipFree(d_index);
	gpu_memory -= n*sizeof(int);

#ifdef DEBUG_MERGE
	hipMemcpy(rn,d_rn,n_work*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(rx,d_rx,n_work*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(ry,d_ry,n_work*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(rmass,d_rmass,n_work*sizeof(double),hipMemcpyDeviceToHost);
	int st=0;
	for( int i=0;i<n_work;i++ ){
		if(rn[i]==0){
		printf("region:%d, %d particles, xcm=%.3f, ycm=%.3f, mass=%.3f\n",st+i,rn[st+i],rx[st+i],ry[st+i],rmass[st+i]);
		}
	}
#endif
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&testtime, start, stop);
#ifdef OUTPUT_DETAIL
	printf("[step2] %.3e(ms) : merge particles in different subregions\n",testtime);
	printf("[step2] GPU memory usage : %d bytes\n",gpu_memory);
#endif 
	time += testtime;
	//===================End of 'Merge' particles in different region=======================


		
	//==================='Merge' particles with global memory====================
	// Calculate memory for shared and global memory
	int gl_node = (pow(nx,2)-1)*4/3+1;
	hipMemcpyToSymbol(HIP_SYMBOL( global_node), &gl_node, sizeof(int));
#ifdef OUTPUT_DETAIL
	printf("[step3] Allocate %d GNODEs\n",gl_node);
#endif
	// Define global & shared memory
	GNODE *root;
	root = (GNODE *)malloc(gl_node*sizeof(GNODE));
	GNODE *d_root;
	hipMalloc((void**)&d_root, gl_node*sizeof(GNODE));
	gpu_memory += gl_node*sizeof(GNODE);
	
	// Calculate the Morton ordering of the sub-regions
	// Align the sub-regions with Morton ordering
	int *morton_index,*d_morton_index;
	morton_index = (int *)malloc(n_work*sizeof(int));
	hipMalloc((void**)&d_morton_index,n_work*sizeof(int));
	gpu_memory += n_work*sizeof(int);
	int which_region = 0;
	block(nx,nx,0,0,morton_index,&which_region);
	hipMemcpy(d_morton_index,morton_index,n_work*sizeof(int),hipMemcpyHostToDevice);
	//printf("Finish calculate morton ordering\n");

	// Call kernel function:
	// Input:center of mass and mass for each subregion, region index
	// Output:information in the shared & global memory
	// Calculate the Morton ordering of the sub-regions
	merge_top1<<<blocks,threads>>>(d_rx,d_ry,d_rmass,d_rn,d_morton_index,d_root);
	dim3 block(1,1);
        dim3 thread(32,32);	
	merge_top2<<<block,thread>>>(d_root);
	hipMemcpy(root,d_root,gl_node*sizeof(GNODE), hipMemcpyDeviceToHost);
	hipFree(d_rx);
	hipFree(d_ry);
	hipFree(d_rmass);
	hipFree(d_morton_index);
	gpu_memory -= n_work*(3*sizeof(double)+sizeof(int));
	
	int test=0;
	for( int i=gl_node-nx*nx;i<gl_node;i++ ){
		test += root[i].num;
		//printf("global=%d,num=%d,xm=%.3f\n",i,root[i].num,root[i].centerofmass[0]);
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&testtime, start, stop);
#ifdef OUTPUT_DETAIL
	printf("[step3] Total particle=%d\n",test);
	printf("[step3] %.3f(ms) : Merge particles in different region with global memroy\n",testtime);
	printf("[step3] GPU memroy usage : %d bytes\n",gpu_memory);
#endif

#ifdef DEBUG_GLOBAL
	for( int i=0;i<5;i++ ){
		printf("global=%d,num=%d,xm=%.3f\n",i,root[i].num,root[i].centerofmass[0]);
	}
#endif
	//==================End of merge subregions with global memory==================

	//==================Calculate force with global memory=========================
	hipEventRecord(start,0);

	force_gpu<<<blocks,threads>>>(d_root,d_x,d_y,d_mass,d_fx,d_fy,d_V,d_n);
	hipMemcpy(fx,d_fx,n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(fy,d_fy,n*sizeof(double), hipMemcpyDeviceToHost);
	hipFree(d_root);
	gpu_memory -= gl_node*sizeof(GNODE);
	
	for( int i=0;i<n;i++ ){
		if( abs(fx[i]-100)<1e-7 ){
			printf("Buffer is not enough!!!\n");
			printf("%d,%.3f\n",i,fx[i]);
			exit(1);
		}else if( abs(fx[i]-200)<1e-7 ){
			printf("Error exist!!!\n");
			printf("%d,%.3f\n",i,fx[i]);
			exit(1);
		}
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&testtime, start, stop);
#ifdef OUTPUT_DETAIL
	printf("[step4] %.3f(ms) : Calculating global force\n",testtime);
	printf("[step4] GPU memory usage : %d bytes\n",gpu_memory);
#endif
	//========================End of 'Merge' particle with shared and global memory==================
	


	//=============================Do Load Balance=================================
	hipEventRecord(start,0);

	// Record thread index for each region
	int *reg_thread_index,*reg_index;
	reg_index = (int *)malloc(n_work*sizeof(int));
	reg_thread_index = (int *)malloc(n_work*sizeof(int));
	for( int i=0;i<n_work;i++ ){ reg_index[i]=i; }

	// Record number of regions in each thread
	int *thread_num;
	thread_num = (int *)malloc(n_thread*sizeof(int));
	for( int i=0;i<n_thread;i++ ){ thread_num[i]=0; }

	// Do the load balance
#ifdef OUTPUT_DETAIL
	printf("[step5] Use %d regions, each thread takes %d \n",n_work,n_work/n_thread);
#endif
	if( n_work/n_thread<1 ){
		printf("[Warning!!!] waste of threads\n");
	}
	if( n_work<n_thread ){
		for( int i=0;i<n_work;i++ ){
			reg_index[i] = i;
			thread_num[i] = 1;
		}
	}else{
		balance(rn,reg_thread_index,thread_num);
		HeapSort(reg_thread_index,reg_index,n_work);
	}
	free(reg_thread_index);
	
	// Check to region is left and cumsum the load of each thread
	int check = thread_num[0];
	for( int i=1;i<n_thread;i++ ){
		check += thread_num[i];
		thread_num[i] += thread_num[i-1];
	}
	if( check != n_work ){
		printf("Error no load balance!!!\n");
		exit(1);
	}

	// Copy the data into GPU	
	int *d_region_index,*d_thread_num;
	hipMalloc((void**)&d_region_index,n_work*sizeof(int));
	hipMalloc((void**)&d_thread_num,n_thread*sizeof(int));
	hipMemcpy(d_region_index,reg_index,n_work*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_thread_num,thread_num,n_thread*sizeof(int),hipMemcpyHostToDevice);
	gpu_memory += (n_work+n_thread)*sizeof(int);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&testtime, start, stop);
#ifdef OUTPUT_DETAIL
	printf("[step5] %.3f(ms) :Load balancing\n",testtime);
	printf("[step5] GPU memory usage : %d bytes\n",gpu_memory);
#endif

	//=====================End fo load balance=======================

	//=====================Allocate each particles===================	
	hipEventRecord(start,0);
	// Cumsum of the # of particle in each region
	for( int i=1;i<n_work;i++ ){
		rn[i] += rn[i-1];
	}
	hipMemcpy(d_rn,rn,n_work*sizeof(unsigned int), hipMemcpyHostToDevice);
	
	// Sort the particle by the region index
	int *particle_index,*d_particle_index;
	particle_index = (int *)malloc(n*sizeof(int));
	hipMalloc((void**)&d_particle_index,n*sizeof(int));
	for( int i=0;i<n;i++ ){
		particle_index[i] = i;
	}
	
	double *d_p;
	hipMalloc((void**)&d_p,n*sizeof(double));

#ifdef DEBUG_SORTING
	printf("Before sorting:\n");
	for( int i=n-1;i>n-15;i-- ){
		printf("%.3f %.3f\n",x[i],y[i]);
	}
#endif
	HeapSort(index,particle_index,n);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&testtime, start, stop);
#ifdef OUTPUT_DETAIL
	printf("[step6] %.3f(ms) : Sorting particles\n",testtime);
#endif

	hipEventRecord(start,0);
	hipMemcpy(d_particle_index,particle_index,n*sizeof(int),hipMemcpyHostToDevice);
	spread_par<<<blocks,threads>>>(d_x,d_p,d_particle_index,d_n);
	hipMemcpy(x,d_p,n*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(d_x,d_p,n*sizeof(double),hipMemcpyDeviceToDevice);
	spread_par<<<blocks,threads>>>(d_y,d_p,d_particle_index,d_n);
	hipMemcpy(y,d_p,n*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(d_y,d_p,n*sizeof(double),hipMemcpyDeviceToDevice);
	spread_par<<<blocks,threads>>>(d_mass,d_p,d_particle_index,d_n);
	hipMemcpy(mass,d_p,n*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(d_mass,d_p,n*sizeof(double),hipMemcpyDeviceToDevice);
	hipFree(d_p);


#ifdef DEBUG_SORTING
	printf("After sorting:\n");
	for( int i=n-1;i>n-15;i-- ){
		printf("%.3f %.3f\n",x[i],y[i]);
	}
#endif
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&testtime, start, stop);
#ifdef OUTPUT_DETAIL
	printf("[step6] %.3f(ms) : Locating particles\n",testtime);
#endif
	//====================End of allocate each particles=====================*/


	
	//=====================Calculate force by n-body============================
	hipEventRecord(start,0);
#ifdef DEBUG_FORCE
	for( int i=n-1;i>n-20;i-- ){
		printf("particle id = %d,fx=%.3f\n",i,fx[i]);
	}
#endif
	treeforce<<<blocks,threads>>>(d_x,d_y,d_mass,d_fx,d_fy,d_V,d_rn,d_region_index,d_thread_num,d_n);

#ifdef DEBUG_FORCE
	hipMemcpy(fx,d_fx,n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(V,d_V,sizeof(double), hipMemcpyDeviceToHost);
	for( int i=n-1;i>n-20;i-- ){
		printf("particle id = %d,fx=%.3f(d=%.3e)\n",i,fx[i],fx[i]-fy[i]);
	}
#endif
	hipFree(d_rn);
	hipFree(d_region_index);
	hipFree(d_thread_num);
	gpu_memory -= n_work*sizeof(int)+n_work*sizeof(int)+n_thread*sizeof(int);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&testtime, start, stop);
#ifdef OUTPUT_DETAIL
	printf("[step7] %.3f(ms) : Evaluate force for every subregion\n",testtime);
	printf("[step7] GPU memory usage : %d bytes\n",gpu_memory);
#endif

	//=====================Calculate force by n-body=============================

	
	//=====================Update velocity,position==============================
	hipEventRecord(start,0);
	double *d_v;
	hipMalloc((void**)&d_v,n*sizeof(double));
	hipMalloc((void**)&d_p,n*sizeof(double));
	hipMemcpy(d_v,vx,n*sizeof(double),hipMemcpyHostToDevice);
	spread_par<<<blocks,threads>>>(d_v,d_p,d_particle_index,d_n);
	hipMemcpy(vx,d_p,n*sizeof(double),hipMemcpyDeviceToHost);
	
	hipMemcpy(d_v,vy,n*sizeof(double),hipMemcpyHostToDevice);
	spread_par<<<blocks,threads>>>(d_v,d_p,d_particle_index,d_n);
	hipMemcpy(vy,d_p,n*sizeof(double),hipMemcpyDeviceToHost);
	hipFree(d_p);
	hipFree(d_v);
	hipFree(d_particle_index);

	double *d_vx,*d_vy;
	hipMalloc((void**)&d_vx,n*sizeof(double));
	hipMalloc((void**)&d_vy,n*sizeof(double));
	hipMemcpy(d_vx,vx,n*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(d_vy,vy,n*sizeof(double),hipMemcpyHostToDevice);

	double *E;
	E = (double *)malloc(sizeof(double));
	*E = 0.0;
	hipMemcpy(d_Ek,E,sizeof(double),hipMemcpyHostToDevice);
	update_gpu<<<blocks,threads>>>(d_x,d_y,d_mass,d_vx,d_vy,d_fx,d_fy,d_Ek,d_n);
	hipMemcpy(x,d_x,n*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(y,d_y,n*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(vx,d_vx,n*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(vy,d_vy,n*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(V,d_V,sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(E,d_Ek,sizeof(double),hipMemcpyDeviceToHost);
	check_boundary(x,y,mass,&n);

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_fx);
	hipFree(d_fy);
	hipFree(d_vx);
	hipFree(d_vy);
	hipFree(d_mass);
	hipMemcpy(E,d_Ek,sizeof(double),hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&testtime, start, stop);
#ifdef OUTPUT_DETAIL
	printf("[step8] %.3f(ms) : update position & velocity\n",testtime);
#endif
	printf("Total energy = %.3e\n",*E+*V);
	printf("Ek = %.3e V = %.3e\n",*E,*V);
	t = t+dt;
	step += 1;
	}
	hipEventRecord(toc,0);
	hipEventSynchronize(toc);
	hipEventElapsedTime(&testtime,tic,toc);
	printf("Total %d steps, %.3f(s). Average %.3f(s) every step\n",step,testtime/1e3,testtime/1e3/(double)step);

	}// end of [if( argv[1]=="-GPU" )]



	//================================================================//
	//  The following code is for CPU tree implementation
	//===============================================================//
	if( !strcmp(argv[1],"-CPU") ){

	//hipEventRecord(tic,0);
	double t=0.0;
	int step=0;
	int file=0;
	
	char preffix[15] = "./output/snap_";
	char number[5];
	char suffix[5] = ".dat";
	int length;

	float t_tree, t_force, t_update, t_estimate;
	double E,Ek;
	
	while( t<endtime ){
		printf("[Step %d] T=%.3e\n",step,t);
		hipEventRecord(start,0);
		// Create tree
		NODE *head = new NODE();
		create_tree(head, x, y, mass,n);
		//printf("End creating tree...\n");
		if( step == 0 ){
			potential(head,x,y,mass,V,n);
			E = 0;
			for( int i=0;i<n;i++ ){
				E += V[i];
			}
			printf("[CPU]Initial energy:%.3e\n",E);
			head = new NODE();
			create_tree(head,x,y,mass,n);
		}
		hipEventRecord(stop,0);
		hipEventElapsedTime(&t_tree, start, stop);
		printf("[CPU]End creating tree, time=%.5f(ms)\n",t_tree);


		// Calculate force for each particles
		hipEventRecord(start,0);
		force(head, x, y, mass, fx, fy,n);
		//printf("Finish calculating force...\n");
		hipEventRecord(stop,0);
		hipEventElapsedTime(&t_force,start,stop);
		printf("[CPU]End calculating force, time=%.5f(ms)\n",t_force);


		hipEventRecord(start,0);
		update(x,y,vx,vy,n);
		update(vx,vy,fx,fy,n);
		check_boundary(x,y,mass,&n);
		hipEventRecord(stop,0);
		hipEventElapsedTime(&t_update,start,stop);
		printf("[CPU]End updating particle, time=%.5f(ms)\n",t_update);

		// Verification
		//if( step%recstep== ){
			hipEventRecord(start,0);
			potential(head,x,y,mass,V,n);
			//printf("[Step %d] T=%.3e\n",step,t);
			E = 0;
			Ek = 0;
			for( int i=0;i<n;i++ ){
				E += V[i]+0.5*mass[i]*(pow(vx[i],2)+pow(vy[i],2));
				Ek += 0.5*mass[i]*(pow(vx[i],2)+pow(vy[i],2));
			}
			hipEventRecord(stop,0);
			hipEventElapsedTime(&t_estimate,start,stop);
			printf("[CPU]Particle number remains:%d\n",n);
			printf("[CPU]Energy conservation:%.3e\n",E);
			printf("[CPU]Kinetic energy:%.3e\n",Ek);
			printf("[CPU]End estimate energy, time=%.3f(ms)\n",t_estimate);

		//}
		if( step%recstep==0 ){
			//Output snapshots
			sprintf(number,"%d",file);
			length = snprintf(NULL, 0, "%s%s%s",preffix,number,suffix);
			char concated[length+1];
			snprintf(concated,sizeof(concated),"%s%s%s",preffix,number,suffix);
			FILE *outfile;
			outfile = fopen(concated,"w");
			fprintf(outfile, "index\tx\ty\n");
			for( int i=0;i<n;i++ ){
				fprintf(outfile, "%d\t%.3f\t%.3f\n",i,x[i],y[i]);
			}
			printf("[CPU]Record position ...\n");
			fclose(outfile);
			file += 1;
		}

		// Move to next step
		t = t+dt;
		step = step+1;
	}

	//hipEventRecord(toc,0);
	//hipEventSynchronize(toc);
	//hipEventElapsedTime(&testtime,tic,toc);
	//printf("Total %d steps, (s). Average (s) every step\n",step);
	}//end of [if(argv[1]=="-CPU")]
	

	return 0;
}
