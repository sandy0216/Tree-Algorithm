#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <vector>
#include <omp.h>
#include "../inc/init.h"
#include "../inc/def_node.h"
#include "../inc/create_tree.h"
#include "../inc/create_tree_gpu.h"
#include "../inc/force.h"
#include "../inc/print_tree.h"
#include "../inc/tool_main.h"
#include "../inc/param.h"
#include "../inc/heap.h"
#include "../inc/cuapi.h"

using namespace std;


int main( int argc, char* argv[] )
{
	double *x, *y, *mass;
	double *vx, *vy;
	double *fx,*fy;
	double *V;
	double E,Ek;
	double region = 80.0;  // restrict position of the initial particles
	double maxmass = 100.0;

	unsigned long    n  = initial_n;

	double endtime = dt*1;

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	x = (double *)malloc(n*sizeof(double));
	y = (double *)malloc(n*sizeof(double));
	mass = (double *)malloc(n*sizeof(double));
	vx = (double *)malloc(n*sizeof(double));
	vy = (double *)malloc(n*sizeof(double));
	fx = (double *)malloc(n*sizeof(double));
	fy = (double *)malloc(n*sizeof(double));
	V  = (double *)malloc(n*sizeof(double));

	//==================initial conditions========================
	// Create initial conditions
	init(x, y, mass, n, region, boxsize, maxmass);
	for( int i=0;i<n;i++ ){
		vx[i]=vy[i]=0.0;
		//mass[i]=10;
	}
	printf("Finsih creating initial condition...\n");
	
	// Record the initial conditions
	FILE *initfile;
	initfile = fopen("./input/init.dat","w");
	fprintf(initfile, "index\tx\ty\tmass\n");
	for( int i=0;i<n;i++ ){
		fprintf(initfile, "%d\t%.3f\t%.3f\t%.3f\n",i,x[i],y[i],mass[i]);
	}
	fclose(initfile);
	// End of creating intial conditions

	//==================GPU settings==============================
	int gid = 0;
	if( hipSetDevice(gid) != hipSuccess ){
		printf("!!! Cannot select GPU \n");
		exit(1);
	}
	hipSetDevice(gid);

	if( tx*ty>1024 ){
		printf("Number of threads per block must < 1024!!\n");
		exit(0);
	}
	dim3 threads(tx,ty);
	if( bx>65535 || by>65535 ){
		printf("The grid size exceeds the limit!\n");
		exit(0);
	}
	dim3 blocks(bx,by);

	// Set basic parameters of GPU
	int   *d_side,*d_n;
	hipMalloc((void**)&d_side, sizeof(int));
	hipMalloc((void**)&d_n, sizeof(int));
	hipMemcpy(d_side, &nx, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

	double *d_boxsize;
	hipMalloc((void**)&d_boxsize, sizeof(double));
	hipMemcpy(d_boxsize, &boxsize, sizeof(double), hipMemcpyHostToDevice);

	double *d_x,*d_y,*d_mass;
	hipMalloc((void**)&d_x, n*sizeof(double));
	hipMalloc((void**)&d_y, n*sizeof(double));
	hipMalloc((void**)&d_mass, n*sizeof(double));
	hipMemcpy(d_x, x, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, n*sizeof(double), hipMemcpyHostToDevice);

	// Split the particle into different subregion
	// Record the region index of each particle
	int *index, *d_index;   
	index = (int *)malloc(n*sizeof(int));
	hipMalloc((void**)&d_index, n*sizeof(int));	
	// Record number of particle in each region
	unsigned int *regnum, *d_regnum;      
	regnum   = (unsigned int *)malloc(n_work*sizeof(unsigned int));
	hipMalloc((void**)&d_regnum, n_work*sizeof(unsigned int));
	for( int i=0;i<n_work;i++ ){ regnum[i]=0; }
	hipMemcpy(d_regnum, regnum, n_work*sizeof(unsigned int), hipMemcpyHostToDevice);
	// Call kernel function :
	// Input  : parameters, postion fo the particles
	// Output : region index of each particles, number of particles in each region
	split<<<threads,blocks>>>(d_x,d_y,d_index,d_regnum,d_n,d_side,d_boxsize);
	hipMemcpy(regnum,d_regnum,n_work*sizeof(unsigned int),hipMemcpyDeviceToHost);
	hipMemcpy(index,d_index,n*sizeof(int),hipMemcpyDeviceToHost);
	
	// Cumsum of the # of particle in each region
	for( int i=1;i<n_work;i++ ){
		regnum[i] += regnum[i-1];
	}
	hipMemcpy(d_regnum,regnum,n_work*sizeof(unsigned int), hipMemcpyHostToDevice);
	
	// Sort the particle by the region index
	int *particle_index,*d_particle_index;
	particle_index = (int *)malloc(n*sizeof(int));
	hipMalloc((void**)&d_particle_index,n*sizeof(int));
	for( int i=0;i<n;i++ ){
		particle_index[i] = i;
	}
	HeapSort(index,particle_index,n);
	hipMemcpy(d_particle_index,particle_index,n*sizeof(int),hipMemcpyHostToDevice);


	// Define GPU parameters
	NODE *p_local_node;
	p_local_node = (NODE *)malloc(n_work*sizeof(NODE));
	NODE *d_local_node;
	hipMalloc((void**)&d_local_node, n_work*sizeof(NODE));

	tree<<<threads,blocks>>>(d_x,d_y,d_mass,d_particle_index,d_regnum,d_n,d_side,d_boxsize,d_local_node);
	//hipMemcpy(p_local_node,d_local_node, n_work*sizeof(NODE), hipMemcpyDeviceToHost);


	printf("well done\n");






	
	//=================Evolution===============================
	/*double t=0.0;
	int step=0;
	int file=0;
	
	char preffix[15] = "./output/snap_";
	char number[5];
	char suffix[5] = ".dat";
	int length;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float t_tree, t_force, t_update, t_estimate;
	
	while( t<endtime ){
		printf("[Step %d] T=%.3e\n",step,t);
		hipEventRecord(start,0);
		// Create tree
		NODE *head = new NODE();
		create_tree(head, x, y, mass,n);
		//printf("End creating tree...\n");
		if( step == 0 ){
			potential(head,x,y,mass,V,n);
			E = 0;
			for( int i=0;i<n;i++ ){
				E += V[i];
			}
			printf("Initial energy:%.3e\n",E);
		}
		hipEventRecord(stop,0);
		hipEventElapsedTime(&t_tree, start, stop);
		printf("End creating tree, time=%.5f(ms)\n",t_tree);


		// Calculate force for each particles
		hipEventRecord(start,0);
		force(head, x, y, mass, fx, fy,n);
		//printf("Finish calculating force...\n");
		hipEventRecord(stop,0);
		hipEventElapsedTime(&t_force,start,stop);
		printf("End calculating force, time=%.5f(ms)\n",t_force);


		hipEventRecord(start,0);
		update(x,y,vx,vy,n);
		update(vx,vy,fx,fy,n);
		check_boundary(x,y,mass,&n);
		hipEventRecord(stop,0);
		hipEventElapsedTime(&t_update,start,stop);
		printf("End updating particle, time=%.5f(ms)\n",t_update);


		//check_boundary(x,y,mass,&n);

		// Verification
		//if( step%100==0 ){
			hipEventRecord(start,0);
			potential(head,x,y,mass,V,n);
			//printf("[Step %d] T=%.3e\n",step,t);
			E = 0;
			Ek = 0;
			for( int i=0;i<n;i++ ){
				E += V[i]+0.5*mass[i]*(pow(vx[i],2)+pow(vy[i],2));
				Ek += 0.5*mass[i]*(pow(vx[i],2)+pow(vy[i],2));
			}
			hipEventRecord(stop,0);
			hipEventElapsedTime(&t_estimate,start,stop);
			printf("Particle number remains:%d\n",n);
			printf("Energy conservation:%.3e\n",E);
			printf("Kinetic energy:%.3e\n",Ek);
			printf("End estimate energy, time=%.3f(ms)\n",t_estimate);

		//}
	
		if( step%1000==0 ){
			//Output snapshots
			sprintf(number,"%d",file);
			length = snprintf(NULL, 0, "%s%s%s",preffix,number,suffix);
			char concated[length+1];
			snprintf(concated,sizeof(concated),"%s%s%s",preffix,number,suffix);
			FILE *outfile;
			outfile = fopen(concated,"w");
			fprintf(outfile, "index\tx\ty\n");
			for( int i=0;i<n;i++ ){
				fprintf(outfile, "%d\t%.3f\t%.3f\n",i,x[i],y[i]);
			}
			printf("Record position ...\n");
			fclose(outfile);
			file += 1;
		}

		// Move to next step
		t = t+dt;
		step = step+1;
	}*/
	

	return 0;
}
